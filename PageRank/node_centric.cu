#include <stdio.h>
#include <iostream>
#include<vector>
#include<utility>
#include<algorithm>
#include <hip/hip_runtime.h>


using namespace std;



float df = 0.85f;

struct Graph{
    int N; // number of nodes
    int E;
    int *ptrarray;
    int *adjListArray;
    int *outdegree; // contains outdegree of all nodes
    float *pr; // page rank values
};


Graph* buildGraph(vector<pair<int,int>>& edges, int E,int V)
{   
    Graph* G = new Graph();
    G->N = V;
    G->E = E;
    G->ptrarray = new int[V+1];
    G->adjListArray = new int[E]; 
    G-> outdegree = new int[V]();
    G->pr = new float[V]; 

    
    sort(edges.begin(),edges.end(),[](pair<int,int>& e1,pair<int,int>& e2){
        return (e1.first==e2.first)? (e1.second<e2.second): (e1.first<e2.first);
    });
    for(int i=0;i<E;i++){
        G->adjListArray[i] = edges[i].second;
        G->outdegree[edges[i].first]++;
    }

    int x=0;
    for(int i=0;i<=V;i++){
        G->ptrarray[i] = x;
        if(i<V) x+= G->outdegree[i];
    }
    
    return G;
}


Graph* readgraph(const char* file){

    FILE *in_file = fopen(file, "r");

    int E,V = 0;
    fscanf(in_file, "%d %d", &E,&V);

    vector<pair<int,int>> edges(E);
    for (auto& e:edges) {
        fscanf(in_file, "%d %d", &e.first, &e.second);
    }
    fclose(in_file);
    
    return buildGraph(edges, E, V);
} 

// Stores the page rank values of the given Graph structure in output.txt
void storePageRank(Graph* graph,const char* file)
{
    FILE *out_file = fopen(file, "w");
    for (int i=0; i<graph->N; i++) {
        fprintf(out_file, "%f\n", graph->pr[i]);
    }
    fclose(out_file);
}

// Initialises the page rank values of the Graph structure to 1
void initialisePageRank(Graph *graph)
{

    for (int i=0; i<graph->N; i++) {
        graph->pr[i] = 1.0f;
    }
}


__global__ void  UpdatePagerank(const int* ptrArray,const int* adjListArray, 
                                const float* oldpr, float* newpr,int N, float df){

    int idx = blockIdx.x*blockDim.x + threadIdx.x;

    while(idx<N){
        float sum = 0;
        for(int w=ptrArray[idx];w<ptrArray[idx+1];w++){
            int wId = adjListArray[w];
            int wOutDegree = ptrArray[wId+1]-ptrArray[wId];
            sum += oldpr[wId]/wOutDegree;
        }
        newpr[idx] = (df*sum) + (1-df)/N;
        idx += gridDim.x*blockDim.x;
    }
}


void PageRank(Graph* G,int iter,float df,int blocksPerGrid,int threadsPerBlock){


    hipError_t err = hipSuccess;

    cout<<"Initialize arrays in device memory\n";
    
    //Allocate the device ptrArray
    int* d_ptrArray = NULL;
    err = hipMalloc((int **)&d_ptrArray, (G->N+1)*sizeof(int));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector ptrArray (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    //Allocate the device adjListArray
    int* d_adjListArray = NULL;
    err = hipMalloc((int **)&d_adjListArray, G->E*sizeof(int));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector adjListArray (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    //Allocate the device oldpr
    float* d_oldpr = NULL;
    err = hipMalloc((float **)&d_oldpr, G->N*sizeof(float));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector oldpr (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    //Allocate the device newpr output array
    float* d_newpr = NULL;
    err = hipMalloc((float **)&d_newpr, G->N*sizeof(float));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector newpr (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    //copy input data from host to device
    cout<<"Copy input data from host memory to CUDA device memory\n";
    err = hipMemcpy(d_ptrArray, G->ptrarray, (G->N+1)*sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector ptrArray from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_adjListArray, G->adjListArray,  G->E*sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector adjListArray from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_oldpr, G->pr, G->N*sizeof(float), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector pr from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    // grid and block dimension
    dim3 grid(blocksPerGrid,1,1);
    dim3 block(threadsPerBlock,1,1);
    
    while(iter--){
        // Launch the PageRank Update CUDA Kernel
        UpdatePagerank<<<grid, block>>>(d_ptrArray, d_adjListArray, d_oldpr, 
                                        d_newpr,G->N,df);
        err = hipGetLastError();
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to launch UpdatePageRank kernel (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
        
        //update the oldpr array
        err = hipMemcpy(d_oldpr,d_newpr,G->N*sizeof(float),hipMemcpyDeviceToDevice);
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to copy newpr array from device to device (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
    }



    err = hipMemcpy(G->pr,d_oldpr,G->N*sizeof(float),hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy pr array from device to Host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    //free up allocated memory
    err = hipFree(d_ptrArray);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector ptrArray (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_adjListArray);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector adjListArray (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_oldpr);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector oldpr (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_newpr);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector newpr (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    //Reset the device
    err = hipDeviceReset();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

}


int main(){
    char input_file[] = "input.txt";
    char output_file[] = "nodeCentric_output.txt";
    Graph* G = readgraph(input_file);

    initialisePageRank(G);
    int threadsPerBlock = 256;
    int blocksPerGrid = (G->N+threadsPerBlock-1)/threadsPerBlock;
    PageRank(G,10000,df,blocksPerGrid,threadsPerBlock);
    cout<<"PageRank calculation done!!"<<endl;

    storePageRank(G,output_file);
    return 0;
}